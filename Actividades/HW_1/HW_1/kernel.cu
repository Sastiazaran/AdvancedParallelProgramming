#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void unique_gid_calculation(int * data)
{
    int tid = blockDim.x * threadIdx.y + threadIdx.y;

    int num_threads_in_a_block = blockDim.x * blockDim.y;
    int block_offset = blockIdx.x * num_threads_in_a_block;

    int num_threads_in_a_row = num_threads_in_a_block * gridDim.x;
    int row_offset = num_threads_in_a_row * blockIdx.y;

    int gid = tid + block_offset + row_offset;

    printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid: %d - data : %d \n",
        blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}

//int main() {
//    unique_gid_calculation << < 2, 8 >> > ();
//
//    return 0;
//}

int main()
{
    int arraySize = 16;
    int array_byte_size = sizeof(int) * array_size; 
    int h_data[] = { 10, 20, 30, 40, 50, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10 };

    int * d_data;
    hipMalloc((void**)&d_data, array_byte_size);
    cudaMencpy(d_data, h_data, array_byte_size, cudaMencpyHostToDevice);

    dim3 block(2, 2);
    dim3 grid(2, 3);

    unique_grid_calculation << < grid, block >> > (d_data);
    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}

#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

__global__ void addKernel(float *a, float *b, float *c, int nfil, int ncol){
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int index = idy * ncol + idx;
    
    if(idy < nfil && idx < ncol) {
        int sum = 0;
        for (int k = 0; k < ncol; k++) {
            sum += a[idy * ncol + k] * b[k * ncol + idx];
        }
        c[index] = sum;
    }
    
}

int main(void)
{
    float* A_h, * B_h, * C_h;
    float* A_d, * B_d, * C_d;
    int nfil = 12;
    int ncol = 12;
    int N = nfil * ncol;

    hipEvent_t start, stop;
    float time;

    size_t size = N * sizeof(float);

    A_h = (float *)malloc(size);   
    B_h = (float *)malloc(size);
    C_h = (float *)malloc(size);

    

    for (int i = 0; i < nfil; i++) {
        for (int j = 0; j < ncol; j++) {
            A_h[i*ncol+j] = rand() % 10;
            B_h[i*ncol+j] = rand() % 10;
        }
    }

    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    dim3 block_size(32, 32);
    dim3 numBlocks(1, 1);

    addKernel << <  numBlocks, block_size  >> > (C_d, A_d, B_d, nfil, ncol);

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    printf("\n \nMatriz c: \n");
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            printf("%d", C_h[i * ncol + j]);
        }
        printf("\n");
    }

    free(A_h);
    free(B_h);
    free(C_h);

    hipFree(C_d);
    hipFree(A_d);
    hipFree(B_d);

    return 0;
}


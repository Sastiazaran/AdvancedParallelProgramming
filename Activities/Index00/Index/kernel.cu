#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void print()
{
    int i = threadIdx.x;
    printf("[DEVICE] ThreadIdx.x: %d\n", i);
}

int main()
{
    print << <2, 8 >> > ();

    return 0;
}



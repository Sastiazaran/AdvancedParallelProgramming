#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// Kernel mult
__global__ void matrixMultiply(float* A, float* B, float* C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float sum = 0.0f;
        for (int i = 0; i < width; i++) {
            sum += A[row * width + i] * B[i * width + col];
        }
        C[row * width + col] = sum;
    }
}

// Kernel sum
__global__ void matrixAdd(float* A, float* B, float* C, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    // Config matriz
    const int width = 100;
    const int height = 500;
    const int size = width * height;

    // Alojamiento y asignación de mem HOST
    float* h_A = new float[size];
    float* h_B = new float[size];
    float* h_C = new float[size];

    // Inicializar matrices
    std::srand(static_cast<unsigned>(std::time(0)));
    for (int i = 0; i < size; ++i) {
        h_A[i] = static_cast<float>(std::rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(std::rand()) / RAND_MAX;
    }

    // Alojamiento de memoria
    float* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, size * sizeof(float));
    hipMalloc((void**)&d_B, size * sizeof(float));
    hipMalloc((void**)&d_C, size * sizeof(float));

    // HOST -> DEVICE
    hipMemcpy(d_A, h_A, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size * sizeof(float), hipMemcpyHostToDevice);

    // Config cuadrícula
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (width + blockDim.y - 1) / blockDim.y);

    // Medición del tiempo
    auto start = std::chrono::high_resolution_clock::now();

    // Lanzamiento kernel
    matrixMultiply << <gridDim, blockDim >> > (d_A, d_B, d_C, width);
   /* matrixAdd << <gridDim, blockDim >> > (d_A, d_B, d_C, width);*/

    // Error Management
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        std::cerr << "Error in multiplication: " << hipGetErrorString(hipError_t) << std::endl;
        return -1;
    }

    // DEVICE -> HOST
    hipMemcpy(h_C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Tiempo de ejecución de la multiplicación: " << duration.count() << " secs" << std::endl;

    // Liberar mem DEVICE
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Liberar mem HOST
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
